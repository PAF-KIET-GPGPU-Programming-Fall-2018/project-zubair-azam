using System;
using System.Collections.Generic;
using System.Linq;
using System.Text;
using System.Threading.Tasks;

namespace ConsoleApplication3
{
    class Program
    {
        static void Main(string[] args)
        {

            List<string[]> trans = new List<string[]>();
            List<string[]> pattern = new List<string[]>();
            Dictionary<string, int> itemcount = new Dictionary<string, int>();

            trans.Add(new[] { "1", "2", "3" });
            trans.Add(new[] { "1", "2", "3" });
            trans.Add(new[] { "1", "2", "3" });

            pattern.Add(new[] { "1", "2" });
            pattern.Add(new[] { "2", "3" });
            pattern.Add(new[] { "1,", "3" });

            itemcount.Add("1", 1);
            itemcount.Add("2", 1);
            itemcount.Add("3", 1);



            foreach (var item in pattern)
            {
                string[] seq = item;
                int seqsupport = 0;
                foreach (var row in trans)
                {
                    string[] rowdata = row;
                    int count = 0;
                    for (int i = 0; i < seq.Length; i++)
                    {
                        foreach (var cells in rowdata)
                        {
                            if (cells == seq[i])
                            {
                                count++;
                                break;
                            }
                        }
                    }


                    if (count == seq.Length)
                    {
                        seqsupport++;
                    }
                }
                if(seqsupport>2)
                {
                    //confidence
                    for (int i = 0; i < seq.Length; i++)
                    {
                       
                        Console.WriteLine(seq[i]+"=>"+(seqsupport/itemcount[seq[i]]));
                    }
                }
            }





        }
    }
}
