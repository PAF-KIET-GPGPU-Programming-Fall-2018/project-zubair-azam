#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string>
#include <cstring>
#include <cstdlib>
#include <iostream>
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""
#include <ctime>

using namespace std;
#define size 1024

__global__
void mykernel(int *transbit, int *pattern, int * result, int pl,int tn, int tp, int *bits)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int i= tid*pl;
	int support=0;
	if (i < tp)
	{
		const int bitslength = tn;
		
		for (int i9 = 0; i9 < tn; i9++)
		{
			for (int i2 = 0; i2 < pl; i2++)
			{      int pi = pattern[i+i2];
				bits[i9] *= transbit[pi*tn+i9];
			}
		}
		for(int i2=0;i2 <tn;i2++)
		{ support=support+bits[i2];
 		}
	}
		result[tid]=support;
}
int main()
{

	const int Ntbits=1000;
	int transbit[Ntbits] = {1,1,1,0,1,0,1,1,1,0,1,1,1,0,1,1,0,0,1,0,1,1,1,0,0,0,1,0,1,1,1,1,0,1,0,1,1,0,0,0,0,1,0,1,0,0,1,1,1,0,0,1,1,0,0,0,1,0,0,0,0,0,1,0,0,0,0,0,1,0,1,1,0,1,1,1,1,0,1,0,0,0,1,1,0,1,0,1,0,0,1,1,0,1,1,1,1,0,0,0,1,1,0,1,0,1,1,1,1,1,1,1,0,1,1,0,1,0,0,0,1,1,1,0,0,0,1,0,0,1,1,0,0,0,1,0,1,0,0,1,0,1,0,1,1,0,0,0,0,1,0,0,0,1,1,1,0,0,0,0,0,1,0,1,1,0,1,1,0,1,0,1,0,0,0,0,1,1,0,0,1,0,0,0,1,0,0,0,1,1,0,1,1,1,1,1,0,0,1,1,0,1,0,1,1,1,0,1,0,0,0,1,1,0,0,1,1,0,0,0,1,0,1,0,0,0,1,0,1,1,0,0,1,0,1,0,0,0,1,1,0,1,0,0,1,1,1,1,0,1,1,1,0,0,0,0,1,1,0,0,0,1,1,1,0,1,1,0,0,0,0,0,0,0,0,0,0,0,1,1,1,1,0,0,1,0,0,0,0,0,0,1,0,1,1,1,1,0,0,1,1,0,0,1,0,1,1,1,0,1,1,0,1,0,1,0,1,1,1,0,1,0,0,1,0,1,1,1,1,1,0,1,0,0,1,0,0,0,1,0,1,1,0,0,1,1,0,0,0,1,1,0,1,0,1,1,0,0,1,1,1,1,0,0,1,0,0,0,1,0,1,1,0,1,0,0,0,1,0,0,0,1,1,1,1,1,0,1,0,0,1,1,0,1,0,0,0,1,1,0,1,1,0,1,1,0,0,0,1,1,1,1,0,0,0,0,1,0,0,1,0,0,1,0,0,1,0,1,0,0,1,0,1,0,1,1,0,0,0,1,1,1,1,0,1,0,1,1,0,1,0,0,0,1,0,0,1,0,0,1,0,1,1,1,0,0,1,1,1,1,1,1,0,1,1,0,0,1,1,0,1,0,0,0,0,0,1,1,1,0,1,0,1,0,0,0,0,0,0,0,0,1,1,0,0,0,0,0,0,1,0,0,1,0,1,1,0,0,1,0,0,1,0,1,0,1,0,1,1,1,0,0,0,0,1,0,0,0,0,1,1,1,1,1,1,0,1,0,0,0,0,1,1,1,0,0,1,0,0,0,0,1,1,0,1,0,1,0,1,1,1,0,1,0,0,1,0,0,1,1,0,1,0,0,0,0,0,0,1,0,0,0,0,0,0,0,1,0,0,1,0,0,1,0,1,0,1,1,1,1,0,0,0,0,1,1,0,0,1,0,1,0,1,0,1,0,1,1,1,1,1,1,0,0,1,0,1,1,1,0,1,1,1,0,1,1,1,1,0,0,0,0,0,1,0,1,1,1,1,0,0,0,1,0,0,1,0,0,1,1,0,0,1,1,1,0,1,0,0,1,0,0,0,1,0,0,0,1,1,1,1,0,0,0,0,1,0,1,0,0,1,1,0,1,0,0,1,1,0,0,0,0,0,0,1,0,1,1,1,1,0,1,0,1,1,0,0,1,0,1,0,0,0,0,0,1,1,1,1,0,1,1,0,1,1,1,1,0,1,0,0,0,1,0,0,0,1,1,1,1,0,1,0,0,1,0,1,0,1,0,0,1,0,0,0,1,0,0,0,0,0,1,0,0,1,0,0,1,1,1,0,1,1,1,0,1,0,0,1,1,1,1,0,0,1,0,1,1,0,1,1,0,1,1,1,0,1,0,1,1,1,1,0,1,0,1,0,1,0,0,0,0,0,0,1,1,1,1,1,1,0,1,1,0,0,1,0,0,0,1,1,0,0,1,1,1,0,0,0,0,1,1,0,1,0,1,1,0,0,1,0,0,1,1,1,0,0,0,1,0,0,0,1,1,1,0,1,0,1,1,0,1,0,0,1,0,0,0,1,0,1,0,0,0,0,1,1,0,0,0,1,0,0,0,1,1,0,1,0,0,1,0,0,0,0,1,1,0,0,0,1,1,0,1,0,0,0,0,0,1,0,0,0,0,1,0,0,0,1,1,1,1,0,1,0,1,1,1,0,0,0,0,0,0,1,0,0,1,1,1,1,0,1,0,1,1,0,0,0,0,1,0,1,0,0,1,0,0,0,1,0,0,0,1,1,0,0,0,1,1};
const int transNo =100;
const int patternLength = 3;
int totalpatrn = 2;
int bits[transNo] = {1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1,1};

	const int Nofpatrn = totalpatrn*patternLength;
	int pattern[Nofpatrn] = { 0,1,3,	1,2,3 };
	int results[Nofpatrn] = {};
	
	int *d_transbit;
	int *d_pattern;
	int *d_result;
	int *d_bits;

	hipMalloc(&d_transbit, sizeof(int)*Ntbits);
	hipMalloc(&d_pattern, sizeof(int) * Nofpatrn);
	hipMalloc(&d_result, sizeof(int) * Nofpatrn);
	hipMalloc(&d_bits, sizeof(int) * transNo);
	hipMemcpy(d_transbit, transbit, sizeof(int) * Ntbits, hipMemcpyHostToDevice);
	hipMemcpy(d_pattern, pattern, sizeof(int) * Nofpatrn, hipMemcpyHostToDevice);
	hipMemcpy(d_bits, bits, sizeof(int) * transNo, hipMemcpyHostToDevice);

	mykernel<<<1, totalpatrn>>>(d_transbit, d_pattern, d_result, patternLength,transNo,Nofpatrn,d_bits);
	hipDeviceSynchronize();
	hipMemcpy(results, d_result, sizeof(int) * Nofpatrn, hipMemcpyDeviceToHost);

	printf("GPU result\n");	
	for (int i2 = 0; i2 < transNo; i2++)
	{
		printf("%d", results[i2]);

	}
	printf("\n");
 clock_t begin = clock();
 
	for (int i = 0; i < 2; i++)// till pattern length
	{
		
		for (int i2 = 0; i2 < patternLength; i2++)
		{	int point = 0;
			
			for (int i9 = 0; i9 < transNo; i9++)
			{
							
				bits[i9] = bits[i9] * transbit[point + (pattern[(i+1)*i2] * transNo)];
				point++;
				
			}
			printf("\n");
	}

	}
 	clock_t end = clock();
   double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
 printf("%lf", elapsed_secs);
 
	hipFree(d_transbit);
	hipFree(d_pattern);
	hipFree(d_result);
	return 0;
	
}
